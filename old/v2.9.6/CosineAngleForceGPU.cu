#include "hip/hip_runtime.h"
// Copyright (c) 2009-2019 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.
/*
  T. Murashima @ Tohoku University
 */



#include "CosineAngleForceGPU.cuh"
#include "hoomd/TextureTools.h"

#include <assert.h>

// SMALL a relatively small number
#define SMALL Scalar(0.001)

/*! \file CosineAngleForceGPU.cu
    \brief Defines GPU kernel code for calculating the cosine squared angle forces. Used by
    CosineAngleForceComputeGPU.
*/

//! Kernel for calculating cosine squared angle forces on the GPU
/*! \param d_force Device memory to write computed forces
    \param d_virial Device memory to write computed virials
    \param virial_pitch Pitch of 2D virial array
    \param N number of particles
    \param d_pos device array of particle positions
    \param d_params Parameters for the angle force
    \param box Box dimensions for periodic boundary condition handling
    \param alist Angle data to use in calculating the forces
    \param pitch Pitch of 2D angles list
    \param n_angles_list List of numbers of angles stored on the GPU
*/
extern "C" __global__ void gpu_compute_cosine_angle_forces_kernel(Scalar4* d_force,
								  Scalar* d_virial,
								  const unsigned int virial_pitch,
								  const unsigned int N,
								  const Scalar4 *d_pos,
								  const Scalar2 *d_params,
								  BoxDim box,
								  const group_storage<3> *alist,
								  const unsigned int *apos_list,
								  const unsigned int pitch,
								  const unsigned int *n_angles_list)
    {
      // start by identifying which particle we are to handle
      int idx = blockIdx.x * blockDim.x + threadIdx.x;

      if (idx >= N)
        return;

      // load in the length of the list for this thread (MEM TRANSFER: 4 bytes)
      int n_angles = n_angles_list[idx];

      // read in the position of our b-particle from the a-b-c triplet. (MEM TRANSFER: 16 bytes)
      Scalar4 idx_postype = d_pos[idx];  // we can be either a, b, or c in the a-b-c triplet
      Scalar3 idx_pos = make_scalar3(idx_postype.x, idx_postype.y, idx_postype.z);
      Scalar3 a_pos,b_pos,c_pos; // allocate space for the a,b, and c atom in the a-b-c triplet

      // initialize the force to 0
      Scalar4 force_idx = make_scalar4(Scalar(0.0), Scalar(0.0), Scalar(0.0), Scalar(0.0));

      Scalar fab[3], fcb[3];

      // initialize the virial to 0
      Scalar virial[6];
      for (int i = 0; i < 6; i++)virial[i] = Scalar(0.0);

      // loop over all angles
      for (int angle_idx = 0; angle_idx < n_angles; angle_idx++)
        {
	  group_storage<3> cur_angle = alist[pitch*angle_idx + idx];

	  int cur_angle_x_idx = cur_angle.idx[0];
	  int cur_angle_y_idx = cur_angle.idx[1];
	  int cur_angle_type = cur_angle.idx[2];

	  int cur_angle_abc = apos_list[pitch*angle_idx + idx];

	  // get the a-particle's position (MEM TRANSFER: 16 bytes)
	  Scalar4 x_postype = d_pos[cur_angle_x_idx];
	  Scalar3 x_pos = make_scalar3(x_postype.x, x_postype.y, x_postype.z);
	  // get the c-particle's position (MEM TRANSFER: 16 bytes)
	  Scalar4 y_postype = d_pos[cur_angle_y_idx];
	  Scalar3 y_pos = make_scalar3(y_postype.x, y_postype.y, y_postype.z);

	  if (cur_angle_abc == 0)
            {
	      a_pos = idx_pos;
	      b_pos = x_pos;
	      c_pos = y_pos;
            }
	  if (cur_angle_abc == 1)
            {
	      b_pos = idx_pos;
	      a_pos = x_pos;
	      c_pos = y_pos;
            }
	  if (cur_angle_abc == 2)
            {
	      c_pos = idx_pos;
	      a_pos = x_pos;
	      b_pos = y_pos;
            }

	  // calculate dr for a-b,c-b,and a-c
	  Scalar3 dab = a_pos - b_pos;
	  Scalar3 dcb = c_pos - b_pos;
	  Scalar3 dac = a_pos - c_pos;

	  // apply periodic boundary conditions
	  dab = box.minImage(dab);
	  dcb = box.minImage(dcb);
	  dac = box.minImage(dac);

	  // get the angle parameters (MEM TRANSFER: 8 bytes)
	  Scalar2 params = __ldg(d_params + cur_angle_type);
	  Scalar K = params.x;
	  Scalar t_0 = params.y;

	  Scalar rsqab = dot(dab, dab);
	  Scalar rab = fast::sqrt(rsqab);
	  Scalar rsqcb = dot(dcb, dcb);
	  Scalar rcb = fast::sqrt(rsqcb);

	  Scalar c_abbc = dot(dab, dcb);
	  c_abbc /= rab*rcb;  // cos(t)

	  if (c_abbc >  Scalar(1.0)) c_abbc =  Scalar(1.0);
	  if (c_abbc < -Scalar(1.0)) c_abbc = -Scalar(1.0);

	  // actually calculate the force

	  Scalar s_abbc   = Scalar(0.0); //TM 2021/01/07
	  Scalar cot_abbc = Scalar(0.0); //TM 2021/01/07
	  Scalar c0 = Scalar(1.0);       //TM 2021/01/07
	  Scalar s0 = Scalar(0.0);       //TM 2021/01/07
	
	  if (t_0 > Scalar(0.0)){
	    s_abbc = fast::sqrt(Scalar(1.0)-c_abbc*c_abbc); //TM 2021/01/07
	    if (s_abbc < SMALL) s_abbc = SMALL;             //TM 2021/01/07
	    s_abbc = Scalar(1.0) / s_abbc;                  //TM 2021/01/07
	    cot_abbc = c_abbc / s_abbc;                     //TM 2021/01/07
	    c0=fast::cos(t_0);                              //TM 2021/01/07
	    s0=fast::sin(t_0);                              //TM 2021/01/07
	  }

	  Scalar dcosth = c_abbc*c0 + s_abbc*s0; //TM 2021/01/07
	  Scalar tk = K*(Scalar(1.0)-dcosth);    //TM 2021/01/07
	  Scalar a = K*(-c0 + cot_abbc * s0);    //TM 2021/01/07
	  Scalar a11 =  a * c_abbc / rsqab;
	  Scalar a12 = -a / (rab * rcb);
	  Scalar a22 =  a * c_abbc / rsqcb;

	  fab[0] = a11*dab.x + a12*dcb.x;
	  fab[1] = a11*dab.y + a12*dcb.y;
	  fab[2] = a11*dab.z + a12*dcb.z;

	  fcb[0] = a22*dcb.x + a12*dab.x;
	  fcb[1] = a22*dcb.y + a12*dab.y;
	  fcb[2] = a22*dcb.z + a12*dab.z;

	  // the rest should be the same as for the harmonic bond
	  // compute 1/3 of the energy, 1/3 for each atom in the angle
	  Scalar angle_eng = tk*Scalar(Scalar(1.0)/Scalar(3.0)); //TM 2021/01/07

	  // upper triangular version of virial tensor
	  Scalar angle_virial[6];
	  angle_virial[0] = Scalar(1./3.)*(dab.x*fab[0] + dcb.x*fcb[0]);
	  angle_virial[1] = Scalar(1./3.)*(dab.y*fab[0] + dcb.y*fcb[0]);
	  angle_virial[2] = Scalar(1./3.)*(dab.z*fab[0] + dcb.z*fcb[0]);
	  angle_virial[3] = Scalar(1./3.)*(dab.y*fab[1] + dcb.y*fcb[1]);
	  angle_virial[4] = Scalar(1./3.)*(dab.z*fab[1] + dcb.z*fcb[1]);
	  angle_virial[5] = Scalar(1./3.)*(dab.z*fab[2] + dcb.z*fcb[2]);


	  if (cur_angle_abc == 0)
            {
	      force_idx.x += fab[0];
	      force_idx.y += fab[1];
	      force_idx.z += fab[2];
            }
	  if (cur_angle_abc == 1)
            {
	      force_idx.x -= fab[0] + fcb[0];
	      force_idx.y -= fab[1] + fcb[1];
	      force_idx.z -= fab[2] + fcb[2];
            }
	  if (cur_angle_abc == 2)
            {
	      force_idx.x += fcb[0];
	      force_idx.y += fcb[1];
	      force_idx.z += fcb[2];
            }

	  force_idx.w += angle_eng;

	  for (int i = 0; i < 6; i++)virial[i] += angle_virial[i];
        }

      // now that the force calculation is complete, write out the result (MEM TRANSFER: 20 bytes)
      d_force[idx] = force_idx;
      for (int i = 0; i < 6; i++)d_virial[i*virial_pitch+idx] = virial[i];
    }

/*! \param d_force Device memory to write computed forces
    \param d_virial Device memory to write computed virials
    \param virial_pitch pitch of 2D virial array
    \param N number of particles
    \param d_pos device array of particle positions
    \param box Box dimensions (in GPU format) to use for periodic boundary conditions
    \param atable List of angles stored on the GPU
    \param pitch Pitch of 2D angles list
    \param n_angles_list List of numbers of angles stored on the GPU
    \param d_params K and t_0 params packed as Scalar2 variables
    \param n_angle_types Number of angle types in d_params
    \param block_size Block size to use when performing calculations

    \returns Any error code resulting from the kernel launch
    \note Always returns hipSuccess in release builds to avoid the hipDeviceSynchronize()

    \a d_params should include one Scalar2 element per angle type. The x component contains K the stiffness constant
    and the y component contains t_0 the equilibrium angle.
*/
hipError_t gpu_compute_cosine_angle_forces(Scalar4* d_force,
					    Scalar* d_virial,
					    const unsigned int virial_pitch,
					    const unsigned int N,
					    const Scalar4 *d_pos,
					    const BoxDim& box,
					    const group_storage<3> *atable,
					    const unsigned int *apos_list,
					    const unsigned int pitch,
					    const unsigned int *n_angles_list,
					    Scalar2 *d_params,
					    unsigned int n_angle_types,
					    int block_size)
{
  assert(d_params);

  static unsigned int max_block_size = UINT_MAX;
  if (max_block_size == UINT_MAX)
    {
      hipFuncAttributes attr;
      hipFuncGetAttributes(&attr, reinterpret_cast<const void*>((const void *))gpu_compute_cosine_angle_forces_kernel);
      max_block_size = attr.maxThreadsPerBlock;
    }

  unsigned int run_block_size = min(block_size, max_block_size);

  // setup the grid to run the kernel
  dim3 grid( N / run_block_size + 1, 1, 1);
  dim3 threads(run_block_size, 1, 1);
  
  // run the kernel
  gpu_compute_cosine_angle_forces_kernel<<< grid, threads>>>(d_force, d_virial, virial_pitch, N, d_pos, d_params, box,
							     atable, apos_list, pitch, n_angles_list);
  return hipSuccess;
}
